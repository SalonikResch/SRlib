
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string>
using namespace std;


__host__ __device__ inline int setBit(unsigned int x, int n){
  return x | ((unsigned int)1 << n);
}

__host__ __device__ inline int clearBit(unsigned int x, int n){
  return x & ~((unsigned int)1 << n);
}


__host__ __device__ inline int bitAt(unsigned int x, int n){
  return (x >> n) & 1;
}

struct instruction {
  string op;
  int qubit1;
  int qubit2;
};

///////////////////////////////// H
__global__
void H(int N, int j, float *a, float *b)
{
  // Thread index
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  // This thread's (amplitude's) bit value at index j
  unsigned int bitAtj = bitAt(tid, j);

  // Index where bit j is 0
  unsigned int bit0 = clearBit(tid, j);
  // Index where bit j is 1
  unsigned int bit1 = setBit(tid, j);

  // Load in zero values first
  float value0 = a[bit0];
  // Then load in one values
  float value1 = a[bit1];

  //if bitAtj is 0, add them, otherwise subtract them
  int sign = pow(-1, bitAtj);
  b[tid] = (value0 + sign*value1 ) / sqrt(2.0);
  // can't sync across blocks, so store to seperate "temp" memory
}

__global__
void H_2(int N, int j, float *a, float *b)
{
  // Thread index
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  // Copy all values
  a[tid] = b[tid];
  // can't sync across blocks, so store to seperate "temp" memory
}

///////////////////////////////// CNOT
__global__
void CNOT_1(int N, int j, int k, float *a, float *b)
{
  // Thread index
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

  int bitAtj = bitAt(tid, j); // control qubit
  int bitAtk = bitAt(tid, k); // target qubit

  if(bitAtj == 1){ // Only take action of control qubit is 1
    int other = tid; // Swap value with index from flipping target qubit
    if(bitAtk == 0){
      other = setBit(other, k);
    }else{
      other = clearBit(other, k);
    }

    float value = a[other];
    b[tid] = value; // store in temp array
  }
}

__global__
void CNOT_2(int N, int j, int k, float *a, float *b){
    // Thread index
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

  int bitAtj = bitAt(tid, j);
  if(bitAtj == 1){
    a[tid] = b[tid]; // copy into amplitudes if something changed
  }
}

///////////////////////////////// CNOT
// Swap local qubit l with non-local qubit n
__global__
void d_swapQubits_1(int N, int l, int n, float *a, float *b){
  // Thread index
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  // This thread's (amplitude's) bit value at index l
  unsigned int bitAtl = (tid >> l) & 1;
  // This thread's (amplitude's) bit value at index n
  unsigned int bitAtn = (tid >> n) & 1;

  // 2 cases where we swap
  if( (bitAtl == 0) && (bitAtn == 1) ){
    unsigned int other = tid;
    other = setBit(other, l);
    other = clearBit(other, n);
    float value = a[other];
    b[tid] = value;
  }else if( (bitAtl == 1) && (bitAtn == 0) ){
    unsigned int other = tid;
    other = clearBit(other, l);
    other = setBit(other, n);
    float value = a[other];
    b[tid] = value;
  }  
}

__global__
void d_swapQubits_2(int N, int l, int n, float *a, float *b){
  // Thread index
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  // This thread's (amplitude's) bit value at index l
  unsigned int bitAtl = (tid >> l) & 1;
  // This thread's (amplitude's) bit value at index n
  unsigned int bitAtn = (tid >> n) & 1;

  // 2 cases where we swap
  if( (bitAtl == 0) && (bitAtn == 1) ){
    a[tid] = a[tid];
  }else if( (bitAtl == 1) && (bitAtn == 0) ){
    a[tid] = b[tid];
  }  
}


bool outOfBounds(int M, instruction *ins){
  return (ins->qubit1 >= M) || (ins->qubit2 >= M);
}


// Fix: The swapping of pointers in this function has to be reflected in main
void swapQubits(int N, int M, int *sigma, instruction *ins,
                int dimGrid, int dimBlock, float *d_a, float *d_b){

  // The qubits that need to be mapped to local qubits
  int j = ins->qubit1;
  int k = ins->qubit2;

  // k could be -1 if there is no second qubit
  int sigK = -1;
  if(k >= 0){
    sigK = sigma[k]; // if not, we want to know where it's mapped
  }

  // Swap j if necessary
  if(sigma[j] >= M){ // Need to swap j, it's non-local
    for(int i=0;i<M;i++){
      // Find a local qubit which is not k
      if( (sigma[i] < M) && (sigma[i] != sigK)){
        d_swapQubits_1<<<dimGrid, dimBlock>>>(N, sigma[i], sigma[j], d_a, d_b);
        d_swapQubits_2<<<dimGrid, dimBlock>>>(N, sigma[i], sigma[j], d_a, d_b);

        // Swap mappings
        int t = sigma[j];
        sigma[j] = sigma[i];
        sigma[i] = t;

        break;
      } 
    }
  }

  // Swap k if necessary
  if((k >= 0) && (sigma[k] >= M)){ // Need to swap k, it's non-local
    for(int i=0;i<M;i++){
      // Find a local qubit which is not j
      if( (sigma[i] < M) && (sigma[i] != sigma[j])){
        d_swapQubits_1<<<dimGrid, dimBlock>>>(N, sigma[i], sigma[k], d_a, d_b);
        d_swapQubits_2<<<dimGrid, dimBlock>>>(N, sigma[i], sigma[k], d_a, d_b);

        // Swap mappings
        int t = sigma[k];
        sigma[k] = sigma[i];
        sigma[i] = t;

        break;
      } 
    }
  }
}


// Matrix multiplication test
int main(void)
{
  // Number of qubits
  int n = 8;
  // Number of amplitudes
  int N = pow(2, n);
  // Number of qubits per device
  int M = 8;

  int nOperations = 8;
  instruction instructions[8] = {
    {"H", 0, -1},
    {"CNOT", 0, 1},
    {"CNOT", 1, 2},
    {"CNOT", 2, 3},
    {"CNOT", 3, 4},
    {"CNOT", 4, 5},
    {"CNOT", 5, 6},
    {"CNOT", 6, 7}
  };

  // Timing variables
  float CPUtime, GPUtransferTime1, GPUtime, GPUtransferTime2;
  hipEvent_t CPUstart, CPUstop, GPUTransfer1Start, GPUTransfer1Stop,
              GPUstart, GPUstop, GPUTransfer2Start, GPUTransfer2Stop;
  hipEventCreate(&CPUstart);
  hipEventCreate(&CPUstop);
  hipEventCreate(&GPUTransfer1Start);
  hipEventCreate(&GPUTransfer1Stop);
  hipEventCreate(&GPUstart);
  hipEventCreate(&GPUstop);
  hipEventCreate(&GPUTransfer2Start);
  hipEventCreate(&GPUTransfer2Stop);

  // Host and device variables for matrices
  float *a, *d_a, *d_b, *d_t, *g;

  // Allocate on host
  a = (float*)malloc(N*sizeof(float));  // amplitudes
  g = (float*)malloc(N*sizeof(float)); // a "golden" comparison

  // Allocate on device
  hipMalloc(&d_a, N*sizeof(float));  // amplitudes, copy 1
  hipMalloc(&d_b, N*sizeof(float));  // amplitudes, copy 2

  // Fill in values for inputs
  a[0] = 1.0f;
  for(int i=1;i<N;i++)
    a[i] = 0;

  // Copy inputs to device
  hipEventRecord(GPUTransfer1Start, 0);
  hipMemcpy(d_a, a, N*sizeof(float), hipMemcpyHostToDevice);
  hipEventRecord(GPUTransfer1Stop, 0);
  hipEventSynchronize(GPUTransfer1Stop);
  hipEventElapsedTime(&GPUtransferTime1, GPUTransfer1Start, GPUTransfer1Stop);

  // Virtual to physical mapping
  int *sigma = (int*)malloc(n*sizeof(int));
  for(int i=0;i<n;i++)
    sigma[i] = i;

  // Determine the gridsize
  int dimBlock = 1024;
  int dimGrid = (N + dimBlock - 1) / dimBlock; // # of blocks in x, round up

  // Simulate on device
  hipEventRecord(GPUstart, 0);
  for(int i=0;i<nOperations;i++){
    //Check qubit locations, swap if necessary
    if(outOfBounds(M, &instructions[i])){
      swapQubits(N, M, sigma, &instructions[i], dimGrid, dimBlock, d_a, d_b);
    }

    // Do the operation
    if( instructions[i].op == "H"){
      int qubit1 = sigma[instructions[i].qubit1];
      H<<<dimGrid, dimBlock>>>(N, qubit1, d_a, d_b);
      hipDeviceSynchronize();
      d_t = d_a; // swap pointers, b is the new a
      d_a = d_b; // because every amplitude was updated
      d_b = d_t;
    }
    if( instructions[i].op == "CNOT"){
      int qubit1 = sigma[instructions[i].qubit1];
      int qubit2 = sigma[instructions[i].qubit2];
      CNOT_1<<<dimGrid, dimBlock>>>(N, qubit1, qubit2, d_a, d_b);
      hipDeviceSynchronize();
      CNOT_2<<<dimGrid, dimBlock>>>(N, qubit1, qubit2, d_a, d_b);
      hipDeviceSynchronize();
    }  
  }
  
  
  hipEventRecord(GPUstop, 0);
  hipEventSynchronize(GPUstop);
  hipEventElapsedTime(&GPUtime, GPUstart, GPUstop);

  // Check for any issues with kernel
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) 
      printf("Error: %s\n", hipGetErrorString(err));
  

  // Copy results back
  hipEventRecord(GPUTransfer2Start, 0);
  hipMemcpy(a, d_a, N*sizeof(float), hipMemcpyDeviceToHost);
  hipEventRecord(GPUTransfer2Stop, 0);
  hipEventSynchronize(GPUTransfer2Stop);
  hipEventElapsedTime(&GPUtransferTime2, GPUTransfer2Start, GPUTransfer2Stop);

  // Compute the total error
  //float totalError = 0.0f;
  for (int i = 0; i < N; i++){
    printf("a[%d] = %.3f\n", i, a[i]);
    //totalError += abs(c[i]-g[i]);
    //printf("g=%f, c=%f\n", g[i], c[i]);
  }
  //printf("Total error: %f\n", totalError);

  hipFree(d_a);
  hipFree(d_b);
  free(a);
  free(g);

  printf("CPU: %.2f ms, GPU: %.2f ms （compute) + %.2f ms transfer\n", 
    CPUtime, GPUtime, GPUtransferTime1 + GPUtransferTime2);
}