
#include <hip/hip_runtime.h>
#include <stdio.h>
#define TILE_SIZE 32

__global__
void mm(int N, float *a, float *b, float *c)
{
  // rows are in x dimenion, cols in y dimension

  // Shared memory to store tiles for temporary reuse
  __shared__ float sA[TILE_SIZE][TILE_SIZE];
  __shared__ float sB[TILE_SIZE][TILE_SIZE];

  // The row and column of this thread (in output matrix c)
  int row = blockIdx.x*blockDim.x + threadIdx.x;
  int col = blockIdx.y*blockDim.y + threadIdx.y;

  // Each thread computes the sum for its index
  float s = 0;

  // Iterate over the number of tiles we need to bring in all relevant elements
  int T = (N + TILE_SIZE - 1) / TILE_SIZE; // number of tiles we need
  // The row of matrix A (does not change)
  int rowA = row; // same as row in c
  // The col of matrix B (does not change)
  int colB = col; // same as col in c

  for (int i=0; i<T; i++){
    // The colums of a will change as we takes tiles further right
    int colA = i*TILE_SIZE + threadIdx.y; 
    // The rows of b will change as we take tiles further down
    int rowB = i*TILE_SIZE + threadIdx.x;

    // Load tiles from matrices a and b
    if( (colA < N) && (rowB < N) ){ // only threads in tile size
      sA[threadIdx.x][threadIdx.y] = a[rowA*N+colA];
      sB[threadIdx.x][threadIdx.y] = b[rowB*N+colB];
    }else{ // Over the edge, fill with 0s
      sA[threadIdx.x][threadIdx.y] = 0;
      sB[threadIdx.x][threadIdx.y] = 0;
    }
    // Have to make sure all threads are synced
    __syncthreads(); // so we know data is ready as we use it

    // Do matrix-multiplication within tile
    for(int j=0;j<TILE_SIZE;j++)
      s += sA[threadIdx.x][j] * sB[i][threadIdx.y]; // dot-product the row and col
  }

  // Store final answers to global memory
  if( (row<N) && (col<N)){
    c[row*N+col] = s;
  }
}

__global__
void madd(int N, float *a, float *b, float *c)
{
  // The row and column of this thread (in output matrix c)
  int row = blockIdx.x*blockDim.x + threadIdx.x;
  int col = blockIdx.y*blockDim.y + threadIdx.y;

  // Store final answers to global memory
  if( (row<N) && (col<N))
    c[row*N+col] = a[row*N+col] + b[row*N+col];
}


// Matrix multiplication test
int main(void)
{
  // Size of the matrix
  int N = 256;
  // Timing variables
  float CPUtime, GPUtransferTime1, GPUtime, GPUtransferTime2;
  hipEvent_t CPUstart, CPUstop, GPUTransfer1Start, GPUTransfer1Stop,
              GPUstart, GPUstop, GPUTransfer2Start, GPUTransfer2Stop;
  hipEventCreate(&CPUstart);
  hipEventCreate(&CPUstop);
  hipEventCreate(&GPUTransfer1Start);
  hipEventCreate(&GPUTransfer1Stop);
  hipEventCreate(&GPUstart);
  hipEventCreate(&GPUstop);
  hipEventCreate(&GPUTransfer2Start);
  hipEventCreate(&GPUTransfer2Stop);

  // Host and device variables for matrices
  float *a, *b, *c, *d_a, *d_b, *d_c, *g;

  // Allocate on host
  a = (float*)malloc(N*N*sizeof(float));
  b = (float*)malloc(N*N*sizeof(float));
  c = (float*)malloc(N*N*sizeof(float));
  g = (float*)malloc(N*N*sizeof(float));

  // Allocate on device
  hipMalloc(&d_a, N*N*sizeof(float)); 
  hipMalloc(&d_b, N*N*sizeof(float));
  hipMalloc(&d_c, N*N*sizeof(float));

  // Fill in values for inputs
  for (int i=0; i<N; i++) {
    for(int j=0; j<N; j++){
      a[i*N+j] = 1.0f;
      b[i*N+j] = 2.0f;
      c[i*N+j] = 0.0f;
      g[i*N+j] = 0.0f;
    }
  }

  // Compute with CPU to check correctness and compare time
  
  hipEventRecord(CPUstart, 0);
  for(int i=0;i<N;i++){
    for(int j=0;j<N;j++){
      for(int k=0;k<N;k++){
        g[i*N+j] += a[i*N+k] * b[k*N+j];
      }
    }
  }
  hipEventRecord(CPUstop, 0);
  hipEventSynchronize(CPUstop);
  hipEventElapsedTime(&CPUtime, CPUstart, CPUstop);

  // Copy inputs to device
  hipEventRecord(GPUTransfer1Start, 0);
  hipMemcpy(d_a, a, N*N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, N*N*sizeof(float), hipMemcpyHostToDevice);
  hipEventRecord(GPUTransfer1Stop, 0);
  hipEventSynchronize(GPUTransfer1Stop);
  hipEventElapsedTime(&GPUtransferTime1, GPUTransfer1Start, GPUTransfer1Stop);

  // Determine the gridsize
  dim3 dimBlock(TILE_SIZE, TILE_SIZE);
  dim3 dimGrid;
  dimGrid.x = (N + dimBlock.x - 1) / dimBlock.x; // # of blocks in x, round up
  dimGrid.y = (N + dimBlock.y - 1) / dimBlock.y; // # of blocks in y, round up
  dimGrid.z = 1;

  // Matrix multiply on device
  hipEventRecord(GPUstart, 0);
  mm<<<dimGrid, dimBlock>>>(N, d_a, d_b, d_c);
  hipDeviceSynchronize();
  hipEventRecord(GPUstop, 0);
  hipEventSynchronize(GPUstop);
  hipEventElapsedTime(&GPUtime, GPUstart, GPUstop);

  // Check for any issues with kernel
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) 
      printf("Error: %s\n", hipGetErrorString(err));
  

  // Copy results back
  hipEventRecord(GPUTransfer2Start, 0);
  hipMemcpy(c, d_c, N*N*sizeof(float), hipMemcpyDeviceToHost);
  hipEventRecord(GPUTransfer2Stop, 0);
  hipEventSynchronize(GPUTransfer2Stop);
  hipEventElapsedTime(&GPUtransferTime2, GPUTransfer2Start, GPUTransfer2Stop);

  // Compute the total error
  float totalError = 0.0f;
  for (int i = 0; i < N*N; i++){
    totalError += abs(c[i]-g[i]);
    //printf("g=%f, c=%f\n", g[i], c[i]);
  }
  printf("Total error: %f\n", totalError);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  free(a);
  free(b);
  free(c);
  free(g);

  printf("CPU: %.2f ms, GPU: %.2f ms （compute) + %.2f ms transfer\n", 
    CPUtime, GPUtime, GPUtransferTime1 + GPUtransferTime2);
}